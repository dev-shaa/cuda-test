#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#include "utils.cu"

#define N 32

int main(int argc, char const *argv[])
{
    int *host_values = (int *)malloc(N * sizeof(int));
    for (int i = 0; i < N; i++)
        host_values[i] = i;

    int gpu_count;
    hipGetDeviceCount(&gpu_count);

    int size = N / gpu_count;
    hipStream_t *streams = (hipStream_t *)malloc(gpu_count * sizeof(hipStream_t));

    for (int i = 0; i < gpu_count; i++)
    {
        hipSetDevice(i);
        hipStreamCreate(&(streams[i]));

        int *dev_values;
        hipMallocAsync(&dev_values, size * sizeof(int), streams[i]);
        hipMemcpyAsync(dev_values, host_values + size * i, size * sizeof(int), hipMemcpyHostToDevice, streams[i]);

        foo<<<1, size, 0, streams[i]>>>(dev_values);

        hipMemcpyAsync(host_values + size * i, dev_values, size * sizeof(int), hipMemcpyDeviceToHost, streams[i]);
        hipFreeAsync(dev_values, streams[i]);
    }

    for (int i = 0; i < gpu_count; i++)
    {
        hipStreamSynchronize(streams[i]);
        hipStreamDestroy(streams[i]);
    }

    for (int i = 0; i < N; i++)
        printf("%d ", host_values[i]);
    printf("\n");

    free(streams);
    free(host_values);

    return 0;
}
