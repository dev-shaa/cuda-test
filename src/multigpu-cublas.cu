#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <cublasXt.h>
#include <assert.h>

#define M 8
#define K M
#define N M

void phpc_gemm_cublas(const double *a, int lda, const double *b, int ldb, double *c, int ldc, int m, int k, int n, int gpu_count)
{
    assert(gpu_count < 32);

    int devices[32];
    cublasXtHandle_t handle;
    double alpha = 1, beta = 1;

    for (size_t i = 0; i < gpu_count; i++)
        devices[i] = i;

    cublasXtCreate(&handle);
    cublasXtDeviceSelect(handle, gpu_count, devices);

    cublasXtDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, m, k, &alpha, b, ldb, a, lda, &beta, c, ldc);
    cublasXtDestroy(handle);
}

int main(int argc, char const *argv[])
{
#define LDA (K + 8)
#define LDB (N + 4)
#define LDC (N + 3)

    int gpu_count;
    hipGetDeviceCount(&gpu_count);

    printf("GPUS: %d\n", gpu_count);

    double a[M * LDA];
    double b[K * LDB];
    double c[M * LDC];

    for (size_t i = 0; i < M; i++)
    {
        for (size_t j = 0; j < K; j++)
            a[i * LDA + j] = i + 1;
    }

    for (size_t i = 0; i < K; i++)
    {
        for (size_t j = 0; j < N; j++)
            b[i * LDB + j] = j + 1;
    }

    for (size_t i = 0; i < M * LDC; i++)
        c[i] = 0;

    printf("A:\n");
    for (size_t i = 0; i < M; i++)
    {
        for (size_t j = 0; j < LDA; j++)
            printf("%.lf ", a[i * LDA + j]);

        printf("\n");
    }

    printf("B:\n");
    for (size_t i = 0; i < K; i++)
    {
        for (size_t j = 0; j < LDB; j++)
            printf("%.lf ", b[i * LDB + j]);

        printf("\n");
    }

    phpc_gemm_cublas(a, LDA, b, LDB, c, LDC, M, K, N, gpu_count);

    printf("C:\n");
    for (size_t i = 0; i < M; i++)
    {
        for (size_t j = 0; j < LDC; j++)
            printf("%.lf ", c[i * LDC + j]);

        printf("\n");
    }

#undef LDC
#undef LDB
#undef LDA

    return 0;
}
