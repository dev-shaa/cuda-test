#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#include "utils.cu"

#define N 32

int main(int argc, char const *argv[])
{
    int *host_values = (int *)malloc(N * sizeof(int));
    for (int i = 0; i < N; i++)
        host_values[i] = i;

    int *dev_values;
    hipFree(NULL);
    hipMalloc(&dev_values, N * sizeof(int));
    hipMemcpy(dev_values, host_values, N * sizeof(int), hipMemcpyHostToDevice);

    foo<<<1, N>>>(dev_values);

    hipDeviceSynchronize();
    hipMemcpy(host_values, dev_values, N * sizeof(int), hipMemcpyDeviceToHost);

    for (int i = 0; i < N; i++)
        printf("%d ", host_values[i]);
    printf("\n");

    hipFree(dev_values);
    free(host_values);

    return 0;
}
