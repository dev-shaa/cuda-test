#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <assert.h>
#include <math.h>

#define COUNT 32

__global__ void foo(float *values)
{
    values[threadIdx.x] += 1;
}

int main(int argc, char const *argv[])
{
    int gpu_count;
    hipGetDeviceCount(&gpu_count);
    assert(gpu_count > 0);

    int N = gpu_count * COUNT;
    float *host_values;
    hipHostMalloc(&host_values, N * sizeof(float), hipHostMallocDefault);
    for (int i = 0; i < N; i++)
        host_values[i] = i;

    hipStream_t streams[8];
    for (int i = 0; i < gpu_count; i++)
    {
        float *dev_values;

        hipSetDevice(i);
        hipStreamCreate(&(streams[i]));
        hipMallocAsync(&dev_values, COUNT * sizeof(float), streams[i]);
        hipMemcpyAsync(dev_values, host_values + i * COUNT, COUNT * sizeof(float), hipMemcpyHostToDevice, streams[i]);

        foo<<<1, COUNT>>>(dev_values);

        hipMemcpyAsync(host_values + i * COUNT, dev_values, COUNT * sizeof(float), hipMemcpyDeviceToHost, streams[i]);
        hipFreeAsync(dev_values, streams[i]);
    }

    for (int i = 0; i < gpu_count; i++)
    {
        hipSetDevice(i);
        hipStreamSynchronize(streams[i]);
        hipStreamDestroy(streams[i]);
    }

    for (int i = 0; i < N; i++)
        printf("%.f ", host_values[i]);

    printf("\n");

    hipHostFree(host_values);

    return 0;
}
