#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <assert.h>
#include <math.h>

__global__ void foo(float *values)
{
    uint id = blockDim.x * blockIdx.x + threadIdx.x;
    values[id] = sin(values[id]);
}

int main(int argc, char const *argv[])
{
    assert(argc > 1);
    int count = atoi(argv[1]);
    assert(count > 0 && count % 32 == 0);

    int gpu_count;
    hipGetDeviceCount(&gpu_count);
    assert(gpu_count > 0);

    int N = gpu_count * count;
    float *host_values;
    hipHostMalloc(&host_values, N * sizeof(float), hipHostMallocDefault);
    for (int i = 0; i < N; i++)
        host_values[i] = i;

    hipStream_t streams[8];
    for (int i = 0; i < gpu_count; i++)
    {
        float *dev_values;

        hipSetDevice(i);
        hipStreamCreate(&(streams[i]));
        hipMallocAsync(&dev_values, count * sizeof(float), streams[i]);
        hipMemcpyAsync(dev_values, host_values + i * count, count * sizeof(float), hipMemcpyHostToDevice, streams[i]);

        foo<<<count / 32, 32>>>(dev_values);

        hipMemcpyAsync(host_values + i * count, dev_values, count * sizeof(float), hipMemcpyDeviceToHost, streams[i]);
        hipFreeAsync(dev_values, streams[i]);
    }

    for (int i = 0; i < gpu_count; i++)
    {
        hipSetDevice(i);
        hipStreamSynchronize(streams[i]);
        hipStreamDestroy(streams[i]);
    }

    for (int i = 0; i < N; i++)
        printf("%.f ", host_values[i]);

    printf("\n");

    hipHostFree(host_values);

    return 0;
}
