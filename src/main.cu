#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <assert.h>
#include <math.h>

#define N 32

__global__ void foo(float *values)
{
    values[threadIdx.x] = threadIdx.x;
}

int main(int argc, char const *argv[])
{
    int gpu_count;
    hipGetDeviceCount(&gpu_count);
    assert(gpu_count > 0);

    int count = N * gpu_count;
    float *host_values;
    hipHostMalloc(&host_values, count * sizeof(float), hipHostMallocDefault);

    hipStream_t streams[8];
    for (int i = 0; i < gpu_count; i++)
    {
        float *dev_values;

        hipSetDevice(i);
        hipStreamCreate(&(streams[i]));
        hipMallocAsync(&dev_values, N * sizeof(float), streams[i]);
        hipMemcpyAsync(dev_values, host_values + i * N, N * sizeof(float), hipMemcpyHostToDevice, streams[i]);

        foo<<<1, N>>>(dev_values);

        hipMemcpyAsync(host_values + i * N, dev_values, N * sizeof(float), hipMemcpyDeviceToHost, streams[i]);
        hipFreeAsync(dev_values, streams[i]);
    }

    for (int i = 0; i < gpu_count; i++)
    {
        hipSetDevice(i);
        hipStreamSynchronize(streams[i]);
        hipStreamDestroy(streams[i]);
    }

    for (int i = 0; i < count; i++)
        printf("%.f ", host_values[i]);
    printf("\n");

    hipHostFree(host_values);

    return 0;
}
