#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define N 32

__global__ void foo(int *values)
{
    values[threadIdx.x] += 1;
}

int main(int argc, char const *argv[])
{
    int *host_values = (int *)malloc(N * sizeof(int));
    for (int i = 0; i < N; i++)
        host_values[i] = i;

    int *dev_values;
    hipMalloc(&dev_values, N * sizeof(int));
    hipMemcpy(dev_values, host_values, N * sizeof(int), hipMemcpyHostToDevice);

    foo<<<1, N>>>(dev_values);

    hipDeviceSynchronize();
    hipMemcpy(host_values, dev_values, N * sizeof(int), hipMemcpyDeviceToHost);

    for (int i = 0; i < N; i++)
        printf("%d ", host_values[i]);
    printf("\n");

    hipFree(dev_values);
    free(host_values);

    return 0;
}
